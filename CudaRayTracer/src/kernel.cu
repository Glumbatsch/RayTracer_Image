#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Structs.h"
#include "IntersectionTests.h"
#include "stb_image_writer.h"

#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <float.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define cudaCall(x) assert(x == hipSuccess)

#define GetIndex() threadIdx.x + blockIdx.x * blockDim.x
#define IsOutOfBounds(id, image) id >= image->width * image->height

__global__ void DebugRaysKernel(World* world)
{
	int a = 1;
	int b = 2;
	int c = a++ + b;
}


__global__ void PrintMaterialsKernel(World* world)
{
	int count = world->materialCount;
	for (int i = 0; i < count; i++)
	{
		Material m = world->materials[i];
		printf("Material %d: \n",i);
		printf("\t Albedo: %1.3f %1.3f %1.3f\n", m.albedo.x, m.albedo.y, m.albedo.z);
		printf("\t EmitColor: %1.3f %1.3f %1.3f\n", m.emitColor.x, m.emitColor.y, m.emitColor.z);
		printf("\t Roughness: %1.3f\n\n", m.roughness);
	}
}

int g_rayCount;
float3* g_deviceImage;
// Device Stuff
World* d_world;
Camera* d_camera;
DeviceImage* d_image;
hiprandState* d_randStates;

void Raytrace(int imageWidth, int imageHeight);
void CudaInit(int imageWidth, int imageHeight);

__global__ void GeneratePrimaryRaysKernel(DeviceImage* image,
	World* world, Camera* camera, int bounces);
__global__ void ComputeIntersectionsKernel(DeviceImage* image,
	World* world);
__global__ void InitCurandKernel(int rayCount, 
	hiprandState* randStates, int seed);
__global__ void ShadeIntersectionsKernel(DeviceImage* image, 
	World* world, hiprandState* randStates);
__global__ void WriteRayColorToImage(DeviceImage* image, 
	World* world);

// float3 to packed ABGR
int PackColor(float3 color)
{
	int r = (int)(255.9f * color.x);
	int g = (int)(255.9f * color.y);
	int b = (int)(255.9f * color.z);

	int result = 0xFF000000; // alpha
	result += (unsigned char)b << 16;
	result += (unsigned char)g << 8;
	result += (unsigned char)r;
	return result;
}

void WriteBMP(int imageHeight, int imageWidth)
{

	float3* fpixels = (float3*)malloc(sizeof(float3) * g_rayCount);
	int* pixels = (int*)malloc(sizeof(int) * g_rayCount);
	cudaCall(hipMemcpy(fpixels, g_deviceImage,
		sizeof(float3) * g_rayCount, hipMemcpyDeviceToHost));

	for (int y = 0; y < imageHeight; y++)
	{
		for (int x = 0; x < imageWidth; x++)
		{
			int id = x + imageWidth * y;
			pixels[id] = PackColor(fpixels[id]);
		}
	}
	// 4 -> ABGR
	stbi_flip_vertically_on_write(true);
	stbi_write_bmp("image.bmp", imageWidth, imageHeight, 4,
		(void*)pixels);
}

int main()
{
	int imageWidth = 720;
	int imageHeight = 480;

	CudaInit(imageWidth, imageHeight);
	printf("Ray casting... ");
	Raytrace(imageWidth, imageHeight);
	printf(" done!\n");
	printf("Writing image to file...");
	WriteBMP(imageHeight, imageWidth);
	printf(" done!\n");
	std::system("start image.bmp");
	return 0;
}

void CudaInit(int imageWidth, int imageHeight)
{
	g_rayCount = imageWidth * imageHeight;

	Material materials[3] = {};
	materials[0].emitColor = make_float3(0.1f, 0.4f, 0.5f);
	materials[1].albedo = make_float3(0.5f, 0.5f, 0.5f);
	materials[1].roughness = 0.75f;
	materials[2].albedo = make_float3(0.7f, 0.5f, 0.3f);
	materials[2].roughness = 0.75f;

	Plane planes[1] = {};

	// x-y plane in the origin
	planes[0].normal = make_float3(0.0f, 0.0f, 1.0f);
	planes[0].d = 0;
	planes[0].materialIndex = 1;

	Sphere spheres[1] = {};
	spheres[0].position = make_float3(0.0f, 0.0f, 0.0f);
	spheres[0].radius = 0.75f;
	spheres[0].materialIndex = 2;

	World w = {};
	w.materialCount = sizeof(materials) / sizeof(materials[0]);
	w.sphereCount = sizeof(spheres)/ sizeof(spheres[0]);
	w.planeCount = sizeof(planes)/sizeof(planes[0]);
	w.rayCount = g_rayCount;

	cudaCall(hipMalloc(&d_world, sizeof(World)));
	cudaCall(hipMalloc(&w.materials, sizeof(materials)));
	cudaCall(hipMalloc(&w.planes, sizeof(planes)));
	cudaCall(hipMalloc(&w.spheres, sizeof(spheres)));
	cudaCall(hipMalloc(&w.rays, sizeof(Ray) * g_rayCount));
	cudaCall(hipMalloc(&w.intersections, 
		sizeof(Intersection) * g_rayCount));


	cudaCall(hipMemcpy(d_world, &w, sizeof(World),
		hipMemcpyHostToDevice));
	cudaCall(hipMemcpy(w.materials, &materials, sizeof(materials),
		hipMemcpyHostToDevice));
	PrintMaterialsKernel << <1, 1 >> > (d_world);
	cudaCall(hipMemcpy(w.planes, &planes, sizeof(planes),
		hipMemcpyHostToDevice));
	cudaCall(hipMemcpy(w.spheres, &spheres, sizeof(spheres),
		hipMemcpyHostToDevice));

	Camera cam = {};
	cam.position = make_float3(0.0f, -10.0f, 1.0f);
	cam.forward = Normalized(cam.position);
	cam.right = Normalized(Cross(make_float3(0.0f, 0.0f, 1.0f),
		cam.forward));
	cam.up = Normalized(Cross(cam.forward, cam.right));

	cudaCall(hipMalloc(&d_camera, sizeof(Camera)));
	cudaCall(hipMemcpy(d_camera, &cam, sizeof(Camera),
		hipMemcpyHostToDevice));

	DeviceImage image = {};
	image.width = imageWidth;
	image.height = imageHeight;
	image.filmWidth = 0.75f;
	image.filmHeight = 0.75f;

	if (image.width > image.height)
	{
		image.filmHeight = image.filmWidth *
			((float)image.height / (float)image.width);
	}
	else if (image.width < image.height)
	{
		image.filmWidth= image.filmHeight *
			((float)image.width / (float)image.height);
	}

	cudaCall(hipMalloc(&image.pixels,
		sizeof(float3) * g_rayCount));
	g_deviceImage = image.pixels;

	cudaCall(hipMalloc(&d_image, sizeof(DeviceImage)));
	cudaCall(hipMemcpy(d_image, &image, sizeof(DeviceImage),
		hipMemcpyHostToDevice));

	cudaCall(hipMalloc(&d_randStates,
		sizeof(hiprandState) * g_rayCount));
}

__global__ void InitCurandKernel(int rayCount, 
	hiprandState* randStates, int seed)
{
	int id = GetIndex();
	if (id >= rayCount) return;

	hiprand_init(seed, id, 0, &randStates[id]);
}

__global__ void GeneratePrimaryRaysKernel(DeviceImage* image,
	World* world, Camera* camera, int bounces)
{
	int id = GetIndex();
	if (IsOutOfBounds(id, image)) return;

	float3 filmCenter = camera->position - camera->forward;

	int pixelX = id % image->width;
	int pixelY = id / image->width;

	float filmX = -1.0f + 2.0f * ((float)pixelX / (float)image->width);
	float filmY = -1.0f + 2.0f * ((float)pixelY / (float)image->height);

	float3 filmP = filmCenter +
		filmX * camera->right * image->filmWidth * 0.5f +
		filmY * camera->up * image->filmHeight * 0.5f;

	Ray r = {};
	r.origin = camera->position;
	r.direction = Normalized(filmP - camera->position);
	r.bounces = bounces;
	r.color = make_float3(1.0f, 1.0f, 1.0f);
	world->rays[id] = r;
}

__global__ void ComputeIntersectionsKernel(DeviceImage* image,
	World* world)
{
	int id = GetIndex();
	if (IsOutOfBounds(id, image))return;

	Ray ray = world->rays[id];
	if (ray.bounces <= 0) return;

	Intersection closestHit = {};
	closestHit.t = FLT_MAX;

	// Test against all planes
	for (int i = 0; i < world->planeCount; i++)
	{
		float t;
		Plane plane = world->planes[i];
		bool isHit = IntersectPlane(ray, plane,t);
		if (isHit && t < closestHit.t)
		{
			closestHit.t = t;
			closestHit.material = plane.materialIndex;
			closestHit.normal = plane.normal;
		}
	}
	// Test against all spheres
	for (int i = 0; i < world->sphereCount; i++)
	{
		float t;
		Sphere sphere = world->spheres[i];
		bool isHit = IntersectSphere(ray, sphere, t);
		if (isHit && t < closestHit.t)
		{
			closestHit.t = t;
			closestHit.material = sphere.materialIndex;
			closestHit.normal = GetSphereNormal(ray,sphere,t);
		}
	}
	world->intersections[id] = closestHit;
}

__global__ void ShadeIntersectionsKernel(DeviceImage* image,
	World* world, hiprandState* randStates)
{
	int id = GetIndex();
	if (IsOutOfBounds(id, image)) return;

	Ray r = world->rays[id];
	if (r.bounces <= 0) return;

	Intersection intersection = world->intersections[id];
	Material mat = world->materials[intersection.material];
	r.color *= (mat.albedo + mat.emitColor);

	Reflect(r, intersection.t, intersection.normal, &randStates[id],
		mat.roughness);
	if (Length2(mat.emitColor) > 0.0f)
	{
		r.bounces = -1;
	}
	r.bounces -= 1;
	world->rays[id] = r;
}

__global__ void WriteRayColorToImage(DeviceImage* image,
	World* world)
{
	int id = GetIndex();
	if (IsOutOfBounds(id, image)) return;

	Ray r = world->rays[id];
	image->pixels[id] = r.color;
}

void Raytrace(int imageWidth, int imageHeight)
{
	// #Note With a higher block size the InitCurandKernel launch will fail because it requires an insane ~6kb stack frame...
	int threadCount = 128;
	int blockCount = imageWidth * imageHeight / threadCount + 1;
	InitCurandKernel << <blockCount, threadCount>> > (g_rayCount, d_randStates, 1234);
	int maxBounces = 8;
	
	GeneratePrimaryRaysKernel << <blockCount, threadCount >> >
		(d_image, d_world, d_camera, maxBounces);

	for (int i = 0; i < maxBounces; i++)
	{
		ComputeIntersectionsKernel<<<blockCount, threadCount>>>
			(d_image, d_world);

		ShadeIntersectionsKernel<<<blockCount,threadCount>>>
			(d_image, d_world, d_randStates);
	}

	WriteRayColorToImage << <blockCount, threadCount >> >
		(d_image, d_world);
}
// #Todo remove unnecessary parameters from kernels (e.g. image)